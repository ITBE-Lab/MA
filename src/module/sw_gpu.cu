#include "hip/hip_runtime.h"
/***********************************************
* # Copyright 2018. Arne Kutzner
* # Contact: Arne Kutzner
* #          kutzner@hanyang.ac.kr
* #
* # GPL 2.0 applies.
* **********************************************/

// NVIDIA card architecture: https://www.anandtech.com/show/3809/nvidias-geforce-gtx-460-the-200-king/2
// https://stackoverflow.com/questions/6647915/cuda-texture-memory-space
// Throw assert code: https://www.softwariness.com/articles/assertions-in-cpp/

#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <iomanip> // std::setw( )
#include <vector>
#include <memory> // for smart pointer
#include <algorithm>
#include <cstring>

#include <util/system.h> // only for measuring runtime

#ifdef __INTELLISENSE__
	#include "hip/hip_runtime_api.h"
#endif

#include "module/sw_gpu.h"
#include "module/sw_gpu_config.h"
#include "module/sw_gpu_defines.h"
#include "module/sw_gpu_kernel.h"

#if ( USE_THREADPOOL == 1)
	#include "util/threadPool.h"
#endif

#if ( DO_TESTS == 1 ) // define in config.h
	#include "../SW_naive.h"
#endif

/* Random nucleotide sequence of length uiLen, represented as codes.
 */
std::vector<char> randomNucleoitdeCodesSeq( const size_t uiLen )
{
	static const char nucleotides[] = { 0, 1, 2, 3 };

	std::vector<char> vNucSeq( uiLen );
	for (size_t i = 0; i < uiLen; ++i)
	{
		vNucSeq[i] = nucleotides[rand() % (sizeof( nucleotides ) - 1)];
	} // for

	return vNucSeq;
} // function

int xScoreMatrix[NUM_OF_SYMBOLS][NUM_OF_SYMBOLS] = { {10, -3, -3, -3},
													 {-3, 10, -3, -3},
													 {-3, -3, 10, -3},
													 {-3, -3, -3, 10} };

/* See http://en.cppreference.com/w/cpp/types/aligned_storage
 */
template<class T, std::size_t N>
class static_vector
{
	// properly aligned uninitialized storage for N T's
	typename std::aligned_storage<sizeof(T), alignof(T)>::type data[N];
	std::size_t m_size = 0;

public:
	// Create an object in aligned storage
	template<typename ...Args> void emplace_back(Args&&... args) 
	{
		if( m_size >= N ) // possible error handling
			throw std::bad_alloc{};
		new(data+m_size) T(std::forward<Args>(args)...);
		++m_size;
	}

	// Access an object in aligned storage
	const T& operator[](std::size_t pos) const 
	{
		// note: needs std::launder as of C++17
		return *reinterpret_cast<const T*>(data+pos);
	}

	// Delete objects from aligned storage
	~static_vector() 
	{
		for(std::size_t pos = 0; pos < m_size; ++pos) {
			// note: needs std::launder as of C++17
			reinterpret_cast<T*>(data+pos)->~T();
		}
	}
}; // class


//// // Texture reference for the query profile.
//// texture<int4, hipTextureType2D, hipReadModeElementType> texRef;

/* Reduce function on device side implementing OR.
 * This version uses contiguous threads, but its interleaved
 * addressing results in many shared memory bank conflicts.
 * For efficiency reasons we use int for boolean values.
 */
__device__ void
reduceOr( int *sdata )
{	/* The outcome is sdata[0]
	 */
	const unsigned int &tid = threadIdx.x;

	__syncthreads();

	// do reduction in shared mem
	for( unsigned int s = 1; s < blockDim.x; s *= 2 )
	{
		int index = 2 * s * tid;

		if( index < blockDim.x )
		{
			sdata[index] = sdata[index] || sdata[index + s];
		} // if

		__syncthreads();
	} // for
} // kernel

/* Reduce function on device side implementing MAX.
 * This version uses contiguous threads, but its interleaved
 * addressing results in many shared memory bank conflicts.
 */
__device__ void
reduceMax( int *sdata )
{	/* The outcome is sdata[0]
	*/
	const unsigned int &tid = threadIdx.x;

	__syncthreads();

	// do reduction in shared mem
	for( unsigned int s = 1; s < blockDim.x; s *= 2 )
	{
		int index = 2 * s * tid;

		if( index < blockDim.x )
		{
			sdata[index] = max(sdata[index], sdata[index + s]);
		} // if

		__syncthreads();
	} // for
} // kernel



/* Non-branching max(x, 0)
 * Instead of branching we have a comparison and multiplication.
 */
__device__ int
limitToZero( int iValue )
{
	//// return (iValue > 0) * iValue;
	return max( iValue, 0 );
} // device function



/* Non-branching maximum.
 * Instead of branching we have a comparison and multiplication.
 * Observation: Slower than standard max
 */
__device__ int
artihmMax( int iFirst, int iSecond )
{
	return iFirst + (iSecond > iFirst) * (iSecond - iFirst);
} // device function

typedef unsigned int int4hw;

/* Per halfword maximum computation*/
__device__ unsigned int
iv2_max( unsigned int uiFirstPair, unsigned int uiSecondPair )
{	/* per-halfword signed comparison: a > b ? 0xffff : 0. */
	auto uiBitVector = __vcmpgts2( uiFirstPair, uiSecondPair );
	return (uiBitVector & uiFirstPair) | (~uiBitVector & uiSecondPair);
}


#if ( SCORES_IN_CONSTANT_MEMORY == 1 )
__constant__ static int4 i4_Scores[256];
#endif

#if ( DO_TESTS == 1 )
//// -- NaiveSW<REFERENCE_SIZE, QUERY_SIZE> pNaiveSW;
#endif


/* Informs about the capabilities of a GPU card */
class GPU_CardInformer {
public:
	const unsigned int uiDeviceId; // DeviceId of the card
	hipDeviceProp_t devProp; // Nvidia properties of the card


	GPU_CardInformer( unsigned int uiDeviceId ) :
		uiDeviceId( uiDeviceId )
	{
		hipGetDeviceProperties( &this->devProp, uiDeviceId );
	} // constructor


	/* Delivers the available free memory for the device */
	size_t freeMemory()
	{	
		hipSetDevice( this->uiDeviceId );
		size_t uiFreeMem, uiTotalMem;
		hipMemGetInfo( &uiFreeMem, &uiTotalMem );
		CUERR

			return uiFreeMem;
	} // method


	/* Prints Info with respect to the card. */
	void printDevInfo()
	{
		std::cout 
			<< "Major revision number:         " << this->devProp.major << "\n" // compute capabilities
			<< "Minor revision number:         " << this->devProp.minor << "\n" // compute capabilities
			<< "Name:                          " << this->devProp.name << "\n"
			<< "Total global memory:           " << this->devProp.totalGlobalMem << "\n"
			<< "Total shared memory per block: " << this->devProp.sharedMemPerBlock << "\n"
			<< "Total registers per block:     " << this->devProp.regsPerBlock << "\n"
			<< "Warp size:                     " << this->devProp.warpSize << "\n"
			<< "Maximum memory pitch:          " << this->devProp.memPitch << "\n"
			<< "Maximum threads per block:     " << this->devProp.maxThreadsPerBlock << "\n"
			<< "Maximum threads per Multiproc.:" << this->devProp.maxThreadsPerMultiProcessor << "\n";
		for( int i = 0; i < 3; ++i )
			std::cout << "Maximum dimension " << i << " of block:   " << this->devProp.maxThreadsDim[i] << "\n";
		for( int i = 0; i < 3; ++i )
			std::cout << "Maximum dimension " << i << " of grid:    " << this->devProp.maxGridSize[i] << "\n";
		std::cout
			<< "Clock rate:                    " << devProp.clockRate << "\n"
			<< "Total constant memory:         " << devProp.totalConstMem << "\n"
			<< "Texture alignment:             " << devProp.textureAlignment << "\n"
			<< "Concurrent copy and execution: " << (devProp.deviceOverlap ? "Yes" : "No") << "\n"
			<< "Number of multiprocessors:     " << devProp.multiProcessorCount << "\n"
			<< "Kernel execution timeout:      " << (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << "\n"
			<< "Number of Concurrent Kernels:  " << devProp.concurrentKernels << "\n"
			<< "Compute Mode                   " << devProp.computeMode << "\n";
	} // method
}; // GPU_CardInformer


template<typename ELEMENT_TYPE>
class AlignedHostVector {
public:
	ELEMENT_TYPE *pHost; // The anchor of the host vector
	size_t uiSize; // size of host array in elements
	size_t uiSizeInBytes; // size of host array in bytes

	/* Constructor */
	AlignedHostVector( size_t uiSize ) :
		uiSize( uiSize ),
		uiSizeInBytes( uiSize * sizeof(ELEMENT_TYPE) )
	{
		hipHostMalloc( (void**)&this->pHost, uiSizeInBytes );
		CUERR
	} // constructor

	/* Delete copy constructor */
	AlignedHostVector(const AlignedHostVector& that) = delete;

	/* Move constructor */
	AlignedHostVector( AlignedHostVector&& rOther ) noexcept 
		: pHost ( rOther.pHost ),
		uiSize( rOther.uiSize )
	{
		std::cout << "AlignedHostVector move constructor" << std::endl;
		pHost = NULL;
		uiSize = 0;
	} // Move Constructor

	  /* Destructor */
	~AlignedHostVector()
	{
		if( pHost != NULL )
			hipHostFree( this->pHost );
	} // destructor
}; // class


/* Wrapper for device vector.
 * Introduction to unified memory: 
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#um-unified-memory-programming-hd
 */
template <typename T_ELEMENT_TYPE>
class DeviceVector {
public:
	const size_t uiCapacity; // size of vector on device in elements

	/* Cache for the device vector */
	std::shared_ptr<std::vector<T_ELEMENT_TYPE>> pvSharedBackup = nullptr;

private:
	const size_t uiCapacityInBytes; // capacity of the vector in bytes
	bool bInUnifiedMemory; // flag that tells about unified memory
	size_t uiBytesUsed; // number of bytes filled by the last update of the vector
	std::unique_ptr<std::vector<T_ELEMENT_TYPE>> pHostCopy; // buffer for an host copy of device vector

	void copyDeviceToHost( void *pDestAddr )
	{
		if( this->bInUnifiedMemory )
		{	/* We can directly copy ... */
			std::memcpy( pDestAddr, this->pvAnchor, this->uiCapacityInBytes );
		} // if
		else
		{	/* We have to copy via hipMemcpyHostToDevice */
			metaMeasureAndLogDuration<false>
				(	"hipMemcpy get device requires",	// text message
					[&] () // lambda by reference
				{
					hipMemcpy( pDestAddr, this->pvAnchor, this->uiCapacityInBytes, hipMemcpyDeviceToHost );
				} // lambda
			); // function call
			CUERR
		} // else
	} // method

	//// /* Indexes are expressed in T_ELEMENT_TYPE */
	//// void copyFromTo( void *pDestAddr, size_t uiStart, size_t uiEnd )
	//// {
	//// 	assert( this->bInUnifiedMemory == false );
	//// 	assert( (uiStart < uiCapacity) && (uiEnd <= uiCapacity) && (uiStart <= uiEnd );
	//// 
	//// 	hipMemcpy( pDestAddr, this->pvAnchor , this->uiCapacityInBytes, hipMemcpyDeviceToHost );
	//// 	CUERR
	//// } // method 

public:
	T_ELEMENT_TYPE* pvAnchor; // anchor of vector on device

	/* Constructor */
	DeviceVector( size_t uiSize, // size of requested vector 
				  bool bAskForUnifiedMemory ) : // use unified memory if available (only available on Linux)
		pvAnchor( NULL ), // anchor of vector on device
		uiCapacity( uiSize ), // size of vector on device in elements
		uiCapacityInBytes( uiSize * sizeof(T_ELEMENT_TYPE) ), // size of vector on device in bytes
		bInUnifiedMemory( false ), // set unified memory flag (later use bAskForUnifiedMemory and check the card properties)
		uiBytesUsed( 0 )
	{
		//// size_t uiFreeMemBefore, uiFreeMemAfter, uiTotalMem;
		//// hipMemGetInfo( &uiFreeMemBefore, &uiTotalMem );
		
		if( bInUnifiedMemory )
		{	/* Allocate unified memory */
			hipMallocManaged( reinterpret_cast<void **>(&pvAnchor), uiCapacityInBytes );
		} // if
		else
		{	/* Allocate standard device memory */
			hipMalloc( reinterpret_cast<void **>(&pvAnchor), uiCapacityInBytes );
		} // else
		CUERR //  TO DO: Raise exception

		/* Continue here be creating this vector if required */
		//// pHostCopy = std::make_unique<std::vector<T_ELEMENT_TYPE>>( this->uiCapacity );

		//// hipMemGetInfo( &uiFreeMemAfter, &uiTotalMem );
		//// std::cout << "RAW: " << uiCapacityInBytes << " Real: " << uiFreeMemBefore - uiFreeMemAfter 
		//// 		  << " DIFF: " << (long)(uiFreeMemBefore - uiFreeMemAfter) - (long)uiCapacityInBytes << std::endl;
	} // constructor

	/* Constructor
	 * Vector is initialized by host-vector.
	 */
	DeviceVector( const std::vector<T_ELEMENT_TYPE> &rvSequence )
		: DeviceVector( rvSequence.size(), true ) // call basic constructor
	{
		this->updateDeviceVector( rvSequence );
	} // constructor

	/* Updates the Device-Vector with the content of the argument vector.
	 * Please note: Normally this is quite expansive, because the vector is not pinned.
	 */
	void updateDeviceVector( const std::vector<T_ELEMENT_TYPE> &rvHostVector )
	{
		assert( rvHostVector.size() <= uiCapacity ); // reject oversized inputs
		
		this->uiBytesUsed = rvHostVector.size() * sizeof( T_ELEMENT_TYPE );
		auto &rvSequenceNoConst = const_cast<std::vector<T_ELEMENT_TYPE> &>(rvHostVector);
		if( this->bInUnifiedMemory )
		{	/* We can directly copy ... */
			std::memcpy( this->pvAnchor, &rvSequenceNoConst[0], this->uiBytesUsed );
		} // if
		else
		{	/* We have to copy via hipMemcpyHostToDevice */
			metaMeasureAndLogDuration<false>
			(	"hipMemcpy update device requires",	// text message
					[&] () // lambda by reference
				{
					hipMemcpy( this->pvAnchor, &rvSequenceNoConst[0], this->uiBytesUsed, hipMemcpyHostToDevice );
				} // lambda
			); // function call
			
		} // else
		CUERR
	} // constructor


	/* Clear the vector on device side */
	void clear()
	{
		hipMemset( this->pvAnchor, 0, this->uiCapacityInBytes );
		CUERR
	} // method

	/* Get a shared pointer to the backup vector */
	inline std::shared_ptr<std::vector<T_ELEMENT_TYPE>> getSharedBackup()
	{
		assert( this->pvSharedBackup != nullptr );
		return this->pvSharedBackup;
	} // method


	/* Update the backup vector */
	void updateSharedBackup()
	{
		if( this->pvSharedBackup == nullptr )
			this->pvSharedBackup = std::make_shared<std::vector<T_ELEMENT_TYPE>>( this->uiCapacity );

		this->copyDeviceToHost( &(*(this->pvSharedBackup))[0] );
	} // method


	/* The Host-vector should be aligned for maximum performance */
	void getCopyIntoVector( std::vector<T_ELEMENT_TYPE> &rvHostVector )
	{
		rvHostVector.resize( this->uiCapacity );
		this->copyDeviceToHost( &rvHostVector[0] );
	} // method


	/* Get a copy of the device-vector on host-side as STL-vector.
	 * This is quite expensive, because the vector must not be pinned.
	 */
	std::vector<T_ELEMENT_TYPE> getCopyAsVector() 
	{	
		std::vector<T_ELEMENT_TYPE> vReturnedVector( this->uiCapacity ); // time expensive
		this->copyDeviceToHost( &vReturnedVector[0] );
		return vReturnedVector;
	} // method


	/* Get a copy of the device-vector on host-side as aligned Host Vector.
	 * Copy for these vectors is a bit faster than for STL-vectors, but allocation takes more time.
	 */
	AlignedHostVector<T_ELEMENT_TYPE> getCopyAsAlignedHostVector()
	{
		AlignedHostVector<T_ELEMENT_TYPE> vReturnedVector( this->uiCapacity ); // time expensive
		this->copyDeviceToHost( vReturnedVector.pHost );
		return vReturnedVector;
	} // method


	/* Get a copy of the device-vector on host-side as array */
	std::unique_ptr<T_ELEMENT_TYPE[]> getCopyAsArray()
	{
		std::unique_ptr<T_ELEMENT_TYPE[]> puArray( new T_ELEMENT_TYPE[this->uiCapacity] );
		auto pArrayOnHost = puArray.get(); // get the inner pointer
		this->copyDeviceToHost( pArrayOnHost );
		return puArray;
	} // method


	/* Copy the content of the complete device vector to a host array */
	void copyToArray( T_ELEMENT_TYPE pArrayOnHost[] )
	{
		this->copyDeviceToHost( pArrayOnHost );
	} // method


	/* Dumps vectors that are holding pairs for debugging purposes. */
	void dump()
	{
		auto vCopyAtHost = this->getCopyAsVector();
		for( auto const &pair : vCopyAtHost )
		{
			std::cout << "(" << pair.x << ", " << pair.y << ") ";
		} // for
		std::cout<< std::endl;
	} // method


	/* Destructor*/
	~DeviceVector()
	{
		if( this->pvAnchor )
		{
			hipFree( this->pvAnchor );
			CUERR
		} // if
	} // destructor
}; // class

/* Segmented device vector on the foundation of pitched CUDA-memory.
 * Here we keep the complete vector on device side.
 */
template <typename T_ELEMENT_TYPE>
class DeviceVector2D {
private:
	uint8_t* pCudaAnchor;
	size_t uiHeight; // height of the 2D matrix
	size_t uiWidth; // width (number of columns) expressed in elements
	size_t uiWidthInBytes; // width expressed in bytes (non-pitched value)
	size_t uiPitch; // expressed in bytes (pitched width)

public:
	DeviceVector2D( size_t uiHeight, size_t uiWidth ) :
		pCudaAnchor( NULL ),
		uiHeight( uiHeight ),
		uiWidth( uiWidth ),
		uiWidthInBytes( uiWidth * sizeof(T_ELEMENT_TYPE) )
	{	/* Allocate pitched memory */
		//// std::cout << "uiHeight " << uiHeight << " uiWidth " << uiWidth << std::endl;
		hipMallocPitch( (void**)&pCudaAnchor, &this->uiPitch, this->uiWidthInBytes, this->uiHeight );
		CUERR
	} // constructor


	/* Delivers device anchor for row with index uiIndex */
	T_ELEMENT_TYPE* operator[] (size_t uiIndex )
	{	
		assert( uiIndex < this->uiHeight );
		return reinterpret_cast<T_ELEMENT_TYPE *>(this->pCudaAnchor + (uiIndex * this->uiPitch));
	} //method


	/* Fill the vector array on the foundation of the argument */
	void fill( const std::vector<T_ELEMENT_TYPE> &rvVector )
	{	/* Check correct size of input vector */
		assert( this->uiWidth * this->uiHeight == rvVector.size() );

		hipMemcpy2D( this->pCudaAnchor, // destination anchor
					  this->uiPitch, // width on device (is device pitch)
					  &rvVector[0], // source anchor
					  this->uiWidthInBytes, // pitch of source memory
					  this->uiWidthInBytes, // width of matrix transfer in bytes
					  this->uiHeight, // height
					  hipMemcpyHostToDevice ); // kind of copy
	} // method


	/* Destructor. Releases all resources */
	~DeviceVector2D()
	{	/* Release allocated CUDA memory. */
		if( pCudaAnchor != NULL )
			hipFree( pCudaAnchor );
		CUERR
	} // destructor
}; // class


/* Query Profile for GPU SW.
 */
template<typename SCORE_TP4, typename SCORE_TP, int STRIPE_WIDTH>
class QueryProfile
{
private:
	/* Correctly casted local copy of scoring matrix */
	SCORE_TP4 xLocalScoreMatrix[NUM_OF_SYMBOLS]; // is aligned to 16 because of TP_4
	
	/* Returns a capacity that is multiple of STRIPE_WIDTH */
	size_t adjustCapacityToStripeWidth( size_t uiSuggestedCapacity )
	{
		assert( uiSuggestedCapacity > 0 );
		auto uiAdjustedCapacity = (((uiSuggestedCapacity - 1) / STRIPE_WIDTH) + 1) * STRIPE_WIDTH;
		return uiAdjustedCapacity;
	} // method

public:
	/* Public attributes */
	const size_t uiCapacity; // capacity of query profile (must be a multiple of STRIPE_WIDTH)

private:
	DeviceVector2D<SCORE_TP4> xDeviceVector; // device vector that keeps the profile
	size_t uiSize; // size of actual profile
	size_t uiNumberOfStripes; // number of stripes in actual profile

public:
	/* Constructor.
	 * uiCapacity is expressed in number of symbols maximally in the query.
	 */
	QueryProfile( const size_t uiSuggestedCapacity ) :
		uiCapacity( adjustCapacityToStripeWidth( uiSuggestedCapacity ) ),
		xDeviceVector( uiCapacity / STRIPE_WIDTH, STRIPE_WIDTH ), // allocated the device vector
		uiSize( 0 ), // profile is initially empty
		uiNumberOfStripes( 0 ) // profile is initially empty
	
	{	/* uiCapacity of stripe must be a multiple of query size */
		assert( this->uiCapacity % STRIPE_WIDTH == 0 );

		/* Initialize local scoring matrix by applying appropriate type cast */
		for( size_t uiRow = 0; uiRow < NUM_OF_SYMBOLS; uiRow++ )
			for( size_t uiColumn = 0; uiColumn < NUM_OF_SYMBOLS; uiColumn++ )
			{
				(reinterpret_cast<SCORE_TP *>(this->xLocalScoreMatrix))[uiRow * NUM_OF_SYMBOLS + uiColumn] 
					= static_cast<SCORE_TP>(xScoreMatrix[uiRow][uiColumn]);
			} // for
	} // constructor

	
	/* Delivers device anchor for row with index uiIndex.
	 * Delivers a pair consisting of a device-vector and stripe-size.
	 */
	std::pair<SCORE_TP4*, size_t> operator[] ( size_t uiStripeIndex )
	{	
		assert( uiStripeIndex < this->uiNumberOfStripes );
		/* Size of requested stripe (of stripe uiIndex) */
		size_t uiRemainder = this->uiSize % STRIPE_WIDTH;
		size_t uiStripeSize = uiStripeIndex >= this->uiNumberOfStripes - 1
									? (uiRemainder == 0 ? STRIPE_WIDTH : uiRemainder) // is last stripe 
									: STRIPE_WIDTH; // is some inner stripe
		assert( uiStripeSize > 0 );
		return std::make_pair( xDeviceVector[uiStripeIndex], uiStripeSize );
	} //method


	inline auto getNumberOfStripes()
	{
		return this->uiNumberOfStripes;
	} // method


	/* Computes the scoring profile on the foundation of the given query.
	 */
	void set( const std::vector<char> &rvQuerySeq )
	{	
		assert( rvQuerySeq.size() <= this->uiCapacity );
		assert( rvQuerySeq.size() > 0 );

		this->uiSize = rvQuerySeq.size();
		this->uiNumberOfStripes = ( ((rvQuerySeq.size() - 1) / STRIPE_WIDTH) + 1 );
		//// std::cout << "rvQuerySeq.size(): " << rvQuerySeq.size() << std::endl;
		//// std::cout << "this->uiNumberOfStripes: " << this->uiNumberOfStripes << std::endl;
		
		/* Compute profile vector and write it to the device */
		std::vector<SCORE_TP4> vHostVector( this->uiCapacity ); // auxiliary host vector
		for( size_t uiIndex = 0; uiIndex < this->uiSize; uiIndex++ )
		{
			vHostVector[uiIndex] = this->xLocalScoreMatrix[rvQuerySeq[uiIndex]];
		} // for
		xDeviceVector.fill( vHostVector );
	} // method
}; // class


//// template<typename SCORE_TP4, typename SCORE_TP2, typename SCORE_TP, int STRIPE_WIDTH>
//// class QueryDescriptor : public QueryProfile<SCORE_TP4, SCORE_TP, STRIPE_WIDTH> {
//// public:
//// 	std::vector<SCORE_TP2> vHostBackup; // Backup of the HE-vector on host side
//// 
//// 
//// 	/* constructor */
//// 	QueryDescriptor( const size_t uiCapacity ) :
//// 		QueryProfile<SCORE_TP4, SCORE_TP, STRIPE_WIDTH>( uiCapacity ),
//// 		vHostBackup( uiCapacity )
//// 	{} // constructor
//// 
//// 
//// 	void storeHE_Vector( DeviceVector<SCORE_TP2> &rxDeviceVector,
//// 						 size_t uiChunkId )
//// 	{	
//// 		assert( uiChunkId < this->uiNumberOfSegments );
//// 		assert( uiChunkId < this->uiNumberOfSegments );
//// 		
//// 		/* You did forget the number of reference segments over here.
//// 		 * We must save the last vector merely, because the inner vector will be uninteresting.
//// 		 */
//// 		//// std::cout << rxDeviceVector.uiCapacity << "   " << this->uiCapacity << std::endl;
//// 		assert( rxDeviceVector.uiCapacity == this->uiCapacity );
//// 
//// 		rxDeviceVector.copyToArray( &vHostBackup[0] );
//// 	} // method
//// }; // class

/* - Can transpose sequence (reference) for efficient GPU-processing
 * - Can pack/unpack sequences
 */
template<typename ELEMENT_TYPE>
class SequenceTransformer {
public:
	/* Copy source to destination and reorganizes the elements according to uiSegmentSize */
	static void copyTransposedTo( const ELEMENT_TYPE aSource[], // copy source
								  ELEMENT_TYPE aDestination[], // copy destination
								  size_t uiChunkSize, // size of chunks
								  size_t uiSourceSize, // size of source
								  size_t uiSegmentSize, // size of segment within chunk
								  size_t uiSourceOffset ) // offset with source
	{	/* Check whether destination size is sound */
		assert( uiChunkSize % uiSegmentSize == 0 ); // is a must!
		assert( uiSourceSize <= uiChunkSize );

		size_t uiNumberOfSegments = uiChunkSize / uiSegmentSize;
		/* TO DO: Could be implemented more efficiently via two nested loops */
		for (size_t uiItr = 0; uiItr < uiChunkSize; uiItr++)
		{
			size_t uiSegmentPos = uiItr / uiNumberOfSegments;
			size_t uiSegmentId = uiItr % uiNumberOfSegments;

			size_t uiRelativeSrcPos = uiSegmentId * uiSegmentSize + uiSegmentPos;
			aDestination[uiItr] = uiRelativeSrcPos < uiSourceSize ? aSource[uiSourceOffset + (uiRelativeSrcPos)] : 0;
		} // for
	} // method


	/* Reorganizes the sequence for optimal memory bandwidth on GPU (coalescing):
	 * TO DO: Do this in place, because it can be done by a simple swapping.
	 * Behavior for segment size 2:
	 * c0, c1, c2, c3, c4, c5, c6, c7 becomes:
	 * c0, c2, c4, c6, c1, c3, c5, c7
	 */
	static std::vector<ELEMENT_TYPE> transposedSeq( const std::vector<ELEMENT_TYPE> &rvSeq, // input vector
												    const size_t uiSegmentSize ) // size of segment within chunk
	{	/* TO DO: Check rvSeq.size() % uiSegmentSize == 0 */
		std::vector<ELEMENT_TYPE> vSegmentedSeq( rvSeq.size() );

		copyTransposedTo( &rvSeq[0], // copy source
						  &vSegmentedSeq[0], // copy destination
						  vSegmentedSeq.size(), // size of chunks
						  vSegmentedSeq.size(),	// length of source
						  uiSegmentSize, // segment size used for reordering
						  0 ); // reference offset

		return vSegmentedSeq;
	} // method


	/* Reverts a transposed vector to its original.
	 * Returns a copy; does not work in place.
	 * TO DO: Could be done in-place by using a juggling approach.
	 */
	static std::vector<ELEMENT_TYPE> inverseTransposedSeq( const std::vector<ELEMENT_TYPE> &rvSeq, // input vector
														   const size_t uiSegmentSize ) // segments size
	{	/* This could be done in-place by using the juggling-principle.
		 * See merging-papers for more information.
		 */
		std::vector<ELEMENT_TYPE> vReturnedSeq( rvSeq.size() );
		size_t uiNumberOfSegments = rvSeq.size() / uiSegmentSize;
		size_t uiRow = 0;
		for( size_t uiSegmentPos = 0; uiSegmentPos < uiSegmentSize; uiSegmentPos++ )
		{	
			for( size_t uiSegmentId = 0; uiSegmentId < uiNumberOfSegments; uiSegmentId++ )
			{
				vReturnedSeq[(uiSegmentId * uiSegmentSize) + uiSegmentPos ] = rvSeq[uiRow++];
			} // for
		} // for

		return vReturnedSeq;
	} // method
}; // class


/* Reference on GPU side.
 * On GPU side references are stored in transposed form in order to use coalescing.
 */
class ChunkedTransposedReference {
private:
	std::vector<std::shared_ptr<std::vector<char>>> vChunks;

public:
	const size_t uiSeqSize; // overall size of the input sequence
	const size_t uiChunkSize; // size of a single chunk consumed by the kernel
	const size_t uiNumOfSegments; // should be equal to BLOCK_SIZE * GRID_SIZE and comes directly form hardware
	const size_t uiNumberOfChunks; // number of big chunks. 
	const size_t uiSegmentSize; // size of segments inside the kernel (used for element transposing)

	/* Constructor */
	ChunkedTransposedReference( std::vector<char> &rvSequence, // input sequence
								const size_t uiChunkSize, // size of the chunks fed to the kernel
								const size_t uiNumOfSegments ) : // should be equal to BLOCK_SIZE * GRID_SIZE and chosen according to hardware
	    /* Prepare the segmented data-structure all chars zero-initialized */
		vChunks(),
		uiSeqSize( rvSequence.size() ),
		uiChunkSize( uiChunkSize ),
		uiNumOfSegments( uiNumOfSegments ),
		//// uiSegmentSize( uiSegmentSize ),
#if ( ALLOW_ABITRARY_REF_SIZE != 1 )
		uiNumberOfPrimarySegments( uiFullSize / uiPrimarySegmentSize ),
#else
		/* Compute number of required chunks */
		//// uiNumberOfChunks( ((uiSeqSize - 1) / uiChunkSize) + 1 ), // old calculation
		uiNumberOfChunks( (uiSeqSize + (uiChunkSize - 1)) / uiChunkSize ),
#endif
		//// uiNumOfSegments( uiChunkSize / uiSegmentSize )
		uiSegmentSize( uiChunkSize / uiNumOfSegments )
		
	{
		assert( uiChunkSize % uiNumOfSegments == 0 ); // this is an absolute must !
		//// assert( uiChunkSize % uiSegmentSize == 0 ); // this is an absolute must !
#if ( ALLOW_ABITRARY_REF_SIZE != 1 )
		assert( uiFullSize % uiPrimarySegmentSize == 0 );
#endif	

		for( size_t uiChunkId = 0; uiChunkId < this->uiNumberOfChunks; uiChunkId++ )
		{	/* Allocate memory for a single chunk.
			 * All chunks are of equal size, there are no undersized chunks 
			 */
			vChunks.push_back( std::make_shared<std::vector<char>>( uiChunkSize ) );
			//// for( auto &cSym : *vChunks.back() )
			//// {
			//// 	cSym = 0;
			//// } // for
			//// std::cout << "COPY SEG:" << uiChunkId << " ADDR: " << size_t(&(*(vChunks[uiChunkId]))[0]) << std::endl;
			SequenceTransformer<char>::copyTransposedTo( &rvSequence[0], // source anchor
														 &(*(vChunks[uiChunkId]))[0], // destination anchor
														 uiChunkSize, // size of chunks
														 numOfSymUsedInChunk( uiChunkId ), // length of source
														 uiSegmentSize, // size of segment within chunk
														 uiChunkId * uiChunkSize ); // offset within source
			
			//// std::cout << "SIZE:" << (*(vChunks[uiChunkId])).size() << std::endl;
			//// for( auto cSymbol : *(vChunks[uiChunkId]) )
			//// {
			//// 	assert( cSymbol >= 0 && cSymbol < NUM_OF_SYMBOLS );
			//// } // for
		} // for
	} // constructor

	/* Delivers the size of the chunk uiChunkId */
	size_t numOfSymUsedInChunk( size_t uiChunkId ) const
	{
		assert( this->uiNumberOfChunks > 0 );
		assert( uiChunkId < this->uiNumberOfChunks );

		if( uiChunkId < (this->uiNumberOfChunks - 1) )
		{	// return size of inner segment
			return this->uiChunkSize;
		} // if
		else
		{	// return size of last segment 
			assert( uiChunkId == uiNumberOfChunks - 1 );
			return this->uiSeqSize % this->uiChunkSize == 0 ? this->uiChunkSize
															: this->uiSeqSize % this->uiChunkSize;
		} // else
	} // method

	std::shared_ptr<std::vector<char>> getChunk( size_t uiIndex ) const
	{
		return vChunks.at( uiIndex ); // we do a range check
	} // method
}; // class

//// /* Encapsulates a reference for SW-kernel purposes */
//// class ReferenceSequenceHolder : public SequenceTransformer<char> {
//// private:
//// 	/* Helper function of the constructor */
//// 	std::vector<char> construct( const std::vector<char> &rvSeq,
//// 								 const size_t uiSegmentSize,
//// 								 bool bPacked )
//// 	{	/* Create a copy of the input-vector */
//// 		std::vector<char> vReturnedSeq( rvSeq );
//// 		
//// 		if( bPacked )
//// 		{	/* Do the packing of the vector */
//// 			pack2Vertically( vReturnedSeq );
//// 		} // of
//// 
//// 		/* Return transposed sequence */
//// 		return transposedSeq( vReturnedSeq, uiSegmentSize );
//// 	} // method
//// 
//// public:
//// 	const std::vector<char> vSequence; // keeps the transposed input sequence
//// 	const bool bPacked; // flag that indicates whether the vector is packed or not
//// 
//// 	/* Constructor */
//// 	ReferenceSequenceHolder( const std::vector<char> &rvSeq, // input sequence
//// 							 const size_t uiSegmentSize, // segment size for coalescing
//// 							 bool bPacked ) : // create packed vector
//// 		vSequence( construct( rvSeq, uiSegmentSize, bPacked ) ),
//// 		bPacked( bPacked )
//// 	{} // constructor
//// }; // class

//// /* Scoring Profile for a single stripe.
////  * The scoring profile should be updated before the stripe kernel is called.
////  */
//// template<typename SCORE_TP4, typename SCORE_TP, int STRIPE_WIDTH>
//// class StripeScoringProfile {
//// public:
//// 	/* (A -> int, C -> int, G -> int, T -> int */
//// 	DeviceVector<SCORE_TP4> xScoresOnDevice; 
//// 	std::vector<SCORE_TP4> vScoresOnHost;
//// 	/* Important observation: Never cast towards vector types like int4, because they are always aligned */
//// 	SCORE_TP4 xLocalScoreMatrix[NUM_OF_SYMBOLS]; // is aligned to 16 !
//// 	
//// 	/* Constructor */
//// 	StripeScoringProfile() :
//// 		xScoresOnDevice( STRIPE_WIDTH, true ), // initialize device vector
//// 		vScoresOnHost( STRIPE_WIDTH ) // initialize host vector
//// 	{	/* Initialize local scoring matrix by applying appropriate type cast */
//// 		for( size_t uiItr = 0; uiItr < NUM_OF_SYMBOLS; uiItr++ )
//// 			for( size_t uiColumn = 0; uiColumn < NUM_OF_SYMBOLS; uiColumn++ )
//// 			{
//// 				(reinterpret_cast<SCORE_TP *>(this->xLocalScoreMatrix))[uiItr * NUM_OF_SYMBOLS + uiColumn] 
//// 					= static_cast<SCORE_TP>(xScoreMatrix[uiItr][uiColumn]);
//// 			} // for
//// 	} // constructor
//// 
//// 	/* Updates the profile on device with the section of the query 
//// 	 * that starts at uiStartIndex 
//// 	 */
//// 	void updateDevice( const std::vector<char> &rvQuerySeq,
//// 					   const size_t uiStartIndex )
//// 	{	/* prepare the profile on host side */
//// 		for( size_t uiColIndex = 0; uiColIndex < STRIPE_WIDTH; uiColIndex++ )
//// 		{
//// 			vScoresOnHost[uiColIndex] = this->xLocalScoreMatrix[rvQuerySeq[uiStartIndex + uiColIndex]];
//// 		} // for
//// 		
//// 		/* Bring the profile to the device.
//// 		 * Update the device vector with the content of the host vector
//// 		 */
//// 		xScoresOnDevice.updateDeviceVector( vScoresOnHost );
//// 	} // method
//// }; // class


/* Works as manager for a collection of device vectors.
 * The class is not managing the device vectors.
 */
template<typename SCORE_TP4, // (int4 for 32 bit, int4 for 2x16 bit
		 typename SCORE_TP2, // (int2 for 32 bit, uint2 for 2x16 bit)
		 typename SCORE_TP, // (int for 32 bit, unsigned int for 2x16 bit)
		 typename CHECKSUM_TP>
class SW_GPU_MemoryCalculator : GPU_CardInformer {
public :
	const size_t uiCostPerRefRow;

	SW_GPU_MemoryCalculator( unsigned int uiDeviceId ) :
		GPU_CardInformer( uiDeviceId ),
		uiCostPerRefRow(   sizeof( SCORE_TP2 ) // HF_VectorOne
						 + sizeof( SCORE_TP2 ) // HF_VectorTwo
						 + sizeof( SCORE_TP ) // M_Vector
						 + sizeof( CHECKSUM_TP ) // C_Vector
						 + sizeof( char ) ) // Reference itself
		
	{} // constructor


	/* Fix-cost independent of any per row-cost.
	 * The alignment-cost per vector can be up to 
	 * Older cards: 1048576
	 * Newer cards: 2097152 
	 */
	size_t fixCost( size_t uiStripeWidth,
					size_t uiNumberOfSegments )
	{
		return   uiStripeWidth * sizeof( SCORE_TP4 ) // Scoring Profile (Query Profile)
				+ uiNumberOfSegments * sizeof( char ) // LazyFixedVector
				+ 2 * uiNumberOfSegments * uiStripeWidth * sizeof( SCORE_TP2 ) // HE_Cache
				+ uiStripeWidth * sizeof( SCORE_TP2 ); // HE_CarryOverVector
	} // method


	/* Computes the maximal number of rows for SW-computation with the current free memory */
	size_t maximalNumberOfRows( size_t uiStripeWidth,
								size_t uiNumberOfSegments )
	{
		auto uiFreeMem = this->freeMemory();
		auto uiFixCost = this->fixCost( uiStripeWidth, uiNumberOfSegments );
		/* 9 vectors are allocated */
		uiFixCost += 2097152 * 9; // estimated maximum costs of all alignments. Older card: 1048576 * 9

		if( uiFreeMem < uiFixCost )
			return 0;

		size_t uiMaxRows = (uiFreeMem - uiFixCost) / uiCostPerRefRow;
		
		/* The number of rows must be a multiple of the uiNumberOfSegments */
		return (uiMaxRows / uiNumberOfSegments) * uiNumberOfSegments;
	} // method
}; // class


 /* Host wrapper for a vector on device.
  * Important in the context of this data-structure is the alignment in the memory.
  * TO DO: Check, that vector sizes on the GPU do not reach beyond the 32-bit world.
  */
template<typename SCORE_TP4, // (int4 for 32 bit, int4 for 2x16 bit
		 typename SCORE_TP2, // (int2 for 32 bit, uint2 for 2x16 bit)
		 typename SCORE_TP, // (int for 32 bit, unsigned int for 2x16 bit)
		 typename CHECKSUM_TP, // 
		 unsigned int STRIP_WIDTH> // width of the strips used for GPU kernel calls
class SW_GPU_Processor : SW_GPU_MemoryCalculator<SCORE_TP4, SCORE_TP2, SCORE_TP, CHECKSUM_TP>
{
public:
	/* This value should be assigned depending on the GPU-hardware.
	 * Typically it is BLOCK_SIZE * GRID_SIZE.
	 * Represents the number of parallel threads.
	 */
	const size_t uiNumberOfSegments;

	/* The reference size is fix after construction. 
	 * The number of segments plus the reference size decide the structure of the outlay reference. 
	 * Warning the value should not be larger than 2^32, or we get problems with the kernel.
	 */
	const size_t uiRefCapicity;
	
	size_t uiSegmentSize; // size of individual segment in device

	/* Device pointer anchors.
	 * Possible improvement: Pitched HE-vector. (The reading of some columns might happen unaligned now.)
	 */
	DeviceVector<SCORE_TP2> xHF_VectorOne; // Size: reference size
	DeviceVector<SCORE_TP2> xHF_VectorTwo; // Size: reference size
	DeviceVector<SCORE_TP> xM_Vector; // Size: reference size
	DeviceVector<CHECKSUM_TP> xC_Vector; // Size: reference size
	DeviceVector<SCORE_TP2> xHE_CarryOverVector; // size: STRIPE_WIDTH
	DeviceVector<SCORE_TP2> xHE_CacheOne; // size: uiNumberOfSegments * STRIPE_WIDTH
	DeviceVector<SCORE_TP2> xHE_CacheTwo; // size: uiNumberOfSegments * STRIPE_WIDTH

	/* Reference vector on device. This vector has size uiRefCapicity.
	 */
	DeviceVector<char> xRefSeqTransposed; 
	
	/* Vector for inter segment communication with respect to the lazy E-loop. 
	 */
	DeviceVector<char> pLazyFixedVector; // size: uiNumberOfSegments
	// Host-copy of the LazyFixedVector
	std::unique_ptr<char[]> pContinuationFlags; // size: uiNumberOfSegments

#if ( USE_THREADPOOL == 1)
	ThreadPool xThreadPool;
	std::future<bool> xMaxExtractSuccess;
#endif

#if ( DO_TESTS == 1 )
	std::vector<char> vRefSeqDebug; // Pointer to reference sequence for debugging
#endif

	/* Clears all device vectors.
	 */
	void clearDeviceVectors()
	{	/* Set all vectors to fully 0 */
		xHF_VectorOne.clear();
		xHF_VectorTwo.clear();
		xHE_CacheOne.clear();
		xHE_CacheTwo.clear();
		xM_Vector.clear();
		xC_Vector.clear();

		pLazyFixedVector.clear();
	} // method

	/* Constructor.
	 * The SW-GPU-Processor gets a reference size as input and tries 
	 * to allocate the required amount of device memory.
	 */
	SW_GPU_Processor( unsigned int uiDeviceId, // device id of GPU
					  size_t uiRequestedSize ) : // requested length of reference sequence
		/* Get the memory calculator initialized */
		SW_GPU_MemoryCalculator<SCORE_TP4, SCORE_TP2, SCORE_TP, CHECKSUM_TP>( uiDeviceId ),

		/* Set the core capacities of the SW-GPU processor */
		uiNumberOfSegments( BLOCK_SIZE * GRID_SIZE ),
		uiRefCapicity( uiRequestedSize > 0 ? uiRequestedSize // 0 indicates user wishes specific size.
									   : this->maximalNumberOfRows( STRIP_WIDTH, uiNumberOfSegments ) ),
		
		/* Segments size is initially 0. This indicates that no reference has been loaded so far */
		uiSegmentSize( 0 ),

		/* Initialize the vectors and scoring profile */
		xHF_VectorOne( uiRefCapicity, true ),
		xHF_VectorTwo( uiRefCapicity, true ),
		xM_Vector( uiRefCapicity, true ),
		xC_Vector( uiRefCapicity, false ),

		xHE_CarryOverVector( STRIP_WIDTH, false ),
		xHE_CacheOne( uiNumberOfSegments * STRIP_WIDTH, true ),
		xHE_CacheTwo( uiNumberOfSegments * STRIP_WIDTH, true ),
		
		xRefSeqTransposed( uiRefCapicity, true ),
		
		/* Initialize the continuation vectors */
		pLazyFixedVector( uiNumberOfSegments, true ),
		pContinuationFlags( new char[this->uiNumberOfSegments] ) // unique pointer
#if ( DO_TESTS == 1 )
		//// , vRefSeqDebug( NULL )
#endif
#if ( USE_THREADPOOL == 1)
		, xThreadPool( 1 )
#endif

	{	/* Reference size must be a multiple of number of segments */
		assert( uiRefCapicity % uiNumberOfSegments == 0 );

		/* Demands the logic of Smith-Waterman's algorithm */
		assert( BLOCK_SIZE >= STRIP_WIDTH );
		
		//// /* The lazy-fixed loop cast from char to long for optimization purposes.
		////  * This cast would be no longer OK if the segment size is no multiple of sizeof(long).
		////  */
		//// assert( uiNumberOfSegments % sizeof( long ) == 0 );

		//// std::cout << "Reference capacity: " << this->uiRefCapicity << std::endl;		
		clearDeviceVectors();
	} // constructor

	/* Load a reference to the SW-GPU processor.
	 * Here we have to deliver a transposed sequence. 
	 */
	void loadReference( const std::vector<char> &rvRefSeq )
	{
		assert( this->uiRefCapicity == rvRefSeq.size() );
		assert( rvRefSeq.size() % this->uiNumberOfSegments == 0 );
		
#if ( DO_TESTS == 1 )
		/* In debug-mode keep a pointer to the loaded reference */
		//// for( auto cSymbol : rvRefSeq )
		//// {
		//// 	assert( cSymbol >= 0 && cSymbol < NUM_OF_SYMBOLS );
		//// } // for

		vRefSeqDebug = SequenceTransformer<char>::inverseTransposedSeq( rvRefSeq, this->uiSegmentSize );
		//// for( auto cSymbol : vRefSeqDebug )
		//// {
		//// 	assert( cSymbol >= 0 && cSymbol < NUM_OF_SYMBOLS );
		//// } // for
		//// vRefSeqDebug = &rvRefSeq;
#endif
		
#if ( USE_PACKED == 1 )
		/* Vector must be packed before transposing */
		pack2Vertically( vReturnedSeq );
#endif

		xRefSeqTransposed.updateDeviceVector( rvRefSeq );
	} // method


	/* Backup function for the "last row" of the active HE-Cache.
	 * Possible Improvement: In the case of unified memory the vector-copy can be avoided.
	 */
	void storeHE_Backup( const size_t uiStripeId, // id of the stripe which receives a backup
						 std::vector<SCORE_TP2> &rvBackupVector, // vector that receives the backup; has size of query
						 DeviceVector<SCORE_TP2>* rxHE_Cache ) // current active HE-cache
	{
		assert( (uiStripeId + 1) * STRIP_WIDTH <= rvBackupVector.size() );
		assert( rxHE_Cache->uiCapacity == STRIP_WIDTH * this->uiNumberOfSegments ); 

		/* Get a host-backup of the device vector */
		std::vector<SCORE_TP2> vHE_CacheCopy = rxHE_Cache->getCopyAsVector();

		size_t uiIndex = this->uiNumberOfSegments - 1; // HE-vector-pitch - 1
		size_t uiOffset = uiStripeId * STRIP_WIDTH;
		for( size_t uiCounter = 0; uiCounter < STRIP_WIDTH; uiCounter++ )
		{
			//// assert( uiOffset + uiCounter < rvBackupVector.size() );
			rvBackupVector[uiOffset + uiCounter] = vHE_CacheCopy[uiIndex];
			uiIndex += uiNumberOfSegments; // HE-vector-pitch
		} // for

		//// /* Get the H-Value of the last row of the HF-vector */
		//// std::vector<SCORE_TP2> vHF_VectorCopy = rxHF_Vector->getCopyAsVector();
		//// rH_ValueBackup = vHF_VectorCopy[vHF_VectorCopy.size() - 1].x;
	} // method


	/* Restores the first row of the HE-vector on the foundation of the backup-vector.
	 * Possible Improvement: In the case of unified memory the vector-copy twice can be avoided.
	 */
	void restoreHE_Backup( const size_t uiStripeId, // id of the stripe which receives a backup
						   const std::vector<SCORE_TP2> &rvBackupVector, // vector that receives the backup; has size of query
						   DeviceVector<SCORE_TP2>* rxHE_CacheSegment0 ) // current active HE-vector
	{
		assert( (uiStripeId + 1) * STRIP_WIDTH <= rvBackupVector.size() );
		assert( rxHE_CacheSegment0->uiCapacity == STRIP_WIDTH );
#if ( 1 )
		/* Do appropriate sub-vector copy */
		auto pFirst = rvBackupVector.begin() + (uiStripeId * STRIP_WIDTH);
		auto pLast = rvBackupVector.begin() + ((uiStripeId + 1) * STRIP_WIDTH);
		std::vector<SCORE_TP2> vStripePart( pFirst, pLast );
		this->xHE_CarryOverVector.updateDeviceVector( vStripePart );

		//// /* Extract the Left-Up-H-Value */
		//// if( uiStripeId == 0 )
		//// {	/* Here we do not have any preceding stripe ... */
		//// 	rLeftUpH_Value = 0;
		//// } // if
		//// else
		//// {
		//// 	auto pFirst = rvBackupVector.begin() + ((uiStripeId - 1) * TILE_WIDTH);
		//// 	auto pLast = rvBackupVector.begin() + (uiStripeId * TILE_WIDTH);
		//// 	std::vector<SCORE_TP2> vStripePart( pFirst, pLast );
		//// 	rLeftUpH_Value = vStripePart[vStripePart.size() - 1].x;
		//// } // else

#else // different HE-vector implementation
		/* Get a host-backup of the device vector.
		 * In the case of unified memory we can avoid this copy.
		 */
		auto vHE_CacheCopy = rxHE_CacheSegment0->getCopyAsVector();

		size_t uiIndex = 0; // We work now with the first row.
		size_t uiOffset = uiStripeId * TILE_WIDTH;
		for( size_t uiCounter = 0; uiCounter < TILE_WIDTH; uiCounter++ )
		{
			vHE_CacheCopy[uiIndex] = rvBackupVector[uiOffset + uiCounter];
			uiIndex += uiNumberOfSegments; // HE-vector-pitch
		} // for

		/* In the case of unified memory we can avoid this copy */
		rxHE_CacheSegment0->updateDeviceVector( vHE_CacheCopy );
#endif
	} // method


	/* The kernel is either in the normal mode or lazy-mode.
	 * The lazy-mode is similar to the lazy F-loop, but works with a checksum.
	 */
	template<bool LAZY_MODE>
	void callTileKernel( bool bHF_VectorsReversed,
						 bool bHE_CacheReversed,
						 SCORE_TP4* pQueryProfileOfStripe, // query profile for stripe
						 SCORE_TP iLeftUpH_Value,
						 const unsigned int uiWidth )  // if FULL_STRIPE_WIDTH is false we get the width of the stripe here 
	{
		dim3 xDimBlock( BLOCK_SIZE, 1 ); // (x, y)
		dim3 xDimGrid( GRID_SIZE, 1 ); // (x, y)
		assert( BLOCK_SIZE * GRID_SIZE * this->uiSegmentSize == this->uiRefCapicity );

		/* Set read and write vector according to the reversed flag */
		auto pvHF_VectorRead = bHF_VectorsReversed ? this->xHF_VectorTwo.pvAnchor : this->xHF_VectorOne.pvAnchor;
		auto pvHF_VectorWrite = bHF_VectorsReversed ? this->xHF_VectorOne.pvAnchor : this->xHF_VectorTwo.pvAnchor;

		auto pvHE_CacheRead = bHE_CacheReversed ? this->xHE_CacheTwo.pvAnchor : this->xHE_CacheOne.pvAnchor;
		auto pvHE_CacheWrite = bHE_CacheReversed ? this->xHE_CacheOne.pvAnchor : this->xHE_CacheTwo.pvAnchor;
		
		/* Call the tile kernel .
		 * In the true branch the compiler unrolls the column iteration in the kernel,
		 * which results in a strong performance boost.
		 */
		if( uiWidth == STRIP_WIDTH )
		{
			tileKernel<SCORE_TP4, SCORE_TP2, SCORE_TP, CHECKSUM_TP, BLOCK_SIZE, STRIP_WIDTH, LAZY_MODE, 
					  true> // call with FULL_STRIPE_WIDTH == true -> This results in unrolling the column iteration 
					  <<<xDimGrid, xDimBlock >>>
				( pvHF_VectorRead, // Vector has reference size
				  pvHF_VectorWrite, // Vector has reference size
				  this->xM_Vector.pvAnchor, // Vector has reference size
				  this->xC_Vector.pvAnchor, // Vector has reference size
				  this->xHE_CarryOverVector.pvAnchor,
				  iLeftUpH_Value,
				  pvHE_CacheRead, // Ingoing row-cache
				  pvHE_CacheWrite, // Outgoing row-cache
				  this->xRefSeqTransposed.pvAnchor, // Address of reference on device
				  pQueryProfileOfStripe, // Query-profile for stripe
				  this->pLazyFixedVector.pvAnchor, // Communicates the need of a continuation
				  (unsigned int)this->uiSegmentSize, // Size of each segment (TILE_HEIGHT)
				  STRIP_WIDTH ); // The argument is only active in the case of FULL_STRIPE_WIDTH is false
		} // if
		else
		{
			tileKernel<SCORE_TP4, SCORE_TP2, SCORE_TP, CHECKSUM_TP, BLOCK_SIZE, STRIP_WIDTH, LAZY_MODE, 
					  false> // call with FULL_STRIPE_WIDTH == false -> Without optimization 
					  <<<xDimGrid, xDimBlock >>>
				( pvHF_VectorRead, // Vector has reference size
				  pvHF_VectorWrite, // Vector has reference size
				  this->xM_Vector.pvAnchor, // Vector has reference size
				  this->xC_Vector.pvAnchor, // Vector has reference size
				  this->xHE_CarryOverVector.pvAnchor,
				  iLeftUpH_Value,
				  pvHE_CacheRead, // Ingoing row-cache
				  pvHE_CacheWrite, // Outgoing row-cache
				  this->xRefSeqTransposed.pvAnchor, // Address of reference on device
				  pQueryProfileOfStripe, // Query-profile for stripe
				  this->pLazyFixedVector.pvAnchor, // Communicates the need of a continuation
				  (unsigned int)this->uiSegmentSize, // Size of each segment (TILE_HEIGHT)
				  uiWidth ); // The argument is only active in the case of FULL_STRIPE_WIDTH is false
		} // else
		CUERR

		/* Change: Work with two maximum-vectors, One active vector and the one of the previous computation.
		 * Use the CPU-time that we wait for hipDeviceSynchronize() for processing the previous maximum vector.
		 */

		/* Wait until the kernel finished its job */
		hipDeviceSynchronize();
	} // method


	 /* Updates all vectors with a fresh symbol of the query.
	  * the resulting kernel call computes a single column of the matrix.
	  * WARNING: Without hipDeviceSynchronize(); we get CUDA-errors.
	  */
	void doStrip( bool bHF_VectorsReversed,
				  const size_t uiStripeId, // id of current stripe
				  std::vector<SCORE_TP2> &rvBackupVector, // HE-backup vector
				  SCORE_TP &riLeftUpH_Value, // second component of backup, the H value-backup
				  SCORE_TP4* pProfileOnDevice, // query profile for stripe
				  const unsigned int uiWidth ) // number of inspected elements 
	{
		assert( uiWidth <= STRIP_WIDTH );
		bool bHE_CacheReversed = false;

		/* initialize incoming HE-vector for the first segment for the current stripe.
		 * Clear the primary HE-vector.
		 * Working with two vectors is a fix, because if we work with one vector we need a pitched data-structure.
		 */
		this->restoreHE_Backup( uiStripeId, rvBackupVector, &(this->xHE_CarryOverVector) ); // current active HE-vector

		this->xHE_CacheOne.clear();

		/* First call of the tile kernel happens in the "init mode" */
		this->callTileKernel<false>( bHF_VectorsReversed, bHE_CacheReversed, pProfileOnDevice, riLeftUpH_Value, uiWidth );

		/* TO DO: Use a counter. Just for paranoid programmers ...*/
		while( true )
		{	/* With each iteration the cache-vectors have to be swapped.
			 * In the beginning the cache vectors have to be cleared.
			 */
			bHE_CacheReversed = !bHE_CacheReversed;

			/* Call the stripe kernel using the lazy-mode, where the checksum decides about continuation */
			this->callTileKernel<true>( bHF_VectorsReversed, bHE_CacheReversed, pProfileOnDevice, riLeftUpH_Value, uiWidth );
	
			if( this->lazyFixed() )
			{
				break;
			} // if
		} // while
#if ( DO_TESTS == 1 )
		//// std::cout << "**** Check Last column:" << std::endl;
		//// this->debugCheckHF_VectorTwo( QUERY_SIZE - 1, bHF_VectorsReversed );
#endif
		/* Save the up-left H-Value for the next call of doStrip */
		riLeftUpH_Value = rvBackupVector[((uiStripeId + 1) * STRIP_WIDTH) - 1].x;
		
		/* Backup the final values of the HE-vector for the current stripe */
		storeHE_Backup( uiStripeId, // id of the stripe which receives a backup
						rvBackupVector, // vector that receives the backup; has size of query
						//// rH_ValueBackup, // H value backup
						bHE_CacheReversed ? &(this->xHE_CacheTwo) : &(this->xHE_CacheOne) ); // current active HE-vector
						//// bHF_VectorsReversed ? &(this->xHF_VectorTwo) : &(this->xHF_VectorOne) ); // current active HF-vector
	} // method


	/* Do the GPU-SW for the given query on the given reference segment.
	 * Changes: Instead of the query deliver the complete query profile.
	 *		    Deliver a reference id, that indicates what reference shall be processed.
	 */
	void doQueryForChunk( QueryProfile<SCORE_TP4, SCORE_TP, STRIP_WIDTH> &rxQueryProfile,
#if ( DO_TESTS == 1 )
							   std::vector<char> &rvQuerySeq, 
#endif
							   std::vector<SCORE_TP2> &rvHE_BackupVector)
							   //// SCORE_TP &rH_ValueBackup, // second component of backup, the H value-backup) // std::vector<char> &rvQuerySeq )
	{	
		/* According to the standard the vector should be zero initialized.
		 * See: https://stackoverflow.com/questions/25198405/are-members-of-structs-in-a-vector-initialized-with-zero-in-c
		 */
		//// ** std::vector<SCORE_TP2> vHE_BackupVector( rvQuerySeq.size() );

#if ( DO_TESTS == 1 )
		/* Fill the SW-matrix for later debugging */
		for( auto cSymbol : this->vRefSeqDebug )
		{
			assert( cSymbol >= 0 && cSymbol < NUM_OF_SYMBOLS );
		} // for
		//// std::shared_ptr< NaiveSW<CHUNK_SIZE, QUERY_SIZE> > pNaiveSW 
		//// 	= std::make_shared<NaiveSW<CHUNK_SIZE, QUERY_SIZE>>();
		//// pNaiveSW->fillMatrix( this->vRefSeqDebug, rvQuerySeq, xScoreMatrix );
#endif

		/* The core loop that iterates over the query in steps of size TILE_WIDTH */
		bool bHF_VectorsReversed = false;
		SCORE_TP iLeftUpH_Value = 0;
		
		for( size_t uiStripeId = 0; uiStripeId < rxQueryProfile.getNumberOfStripes() ; uiStripeId++ )
		{	/* Create the scoring profile in the local vector.
			 * HINT: This is quite expensive.
			 * The profile could be fully prepared by the CPU before calling the GPU.
			 */
			std::pair<SCORE_TP4*, size_t> xStripeData = rxQueryProfile[uiStripeId];
			unsigned int uiStripWidth = static_cast<unsigned int>(xStripeData.second);
			/* Compute matrix for the stripe*/
			this->doStrip( bHF_VectorsReversed,
							uiStripeId, // id of current stripe
							rvHE_BackupVector, // HE-backup vector
							iLeftUpH_Value, // set by reference in method
							xStripeData.first, //// rxQueryProfile.xDeviceVector[uiStripeId],
						    uiStripWidth ); //// TILE_WIDTH ); // number of elements in stripe
			
			/* The HF-vectors must be swap after each call of updateTile */
			bHF_VectorsReversed = !bHF_VectorsReversed;
		} // for

#if ( DO_TESTS == 1 )
		//// std::cout << "**** Check last column:" << std::endl;
		//// /* If the divisor is greater than 1, we get problem here for all segments following the first one */
		//// this->debugCheckHF_VectorTwo( pNaiveSW, QUERY_SIZE - 1, !bHF_VectorsReversed );
#endif
	} // method


	void doQueryForChunkedReference( std::vector<char> &rvQuerySeq, // query
									 const ChunkedTransposedReference &rxChunkedRef, // reference in segmented form
									 std::vector<size_t> &rvMaxScorePositions, // vector receives maximum positions
									 SCORE_TP &iOverallMaxScore )  // vector that logs the maximum-score positions
	{	
		assert( rxChunkedRef.uiChunkSize == this->uiRefCapicity );
		assert( rxChunkedRef.uiSegmentSize == this->uiSegmentSize );
		
		/* Create the query profile */
		QueryProfile<SCORE_TP4, SCORE_TP, STRIP_WIDTH> xQueryProfile( rvQuerySeq.size() );
		xQueryProfile.set( rvQuerySeq );

		/* According to the standard the vector is zero initialized.
		 * See: https://stackoverflow.com/questions/25198405/are-members-of-structs-in-a-vector-initialized-with-zero-in-c
		 */
		std::vector<SCORE_TP2> vHE_BackupVector( xQueryProfile.getNumberOfStripes() * STRIP_WIDTH );

		iOverallMaxScore = 0;

		//// /* Receives and keeps the maximum scores after each kernel call */
		//// std::vector<SCORE_TP> vM_Buffer_Vector;

		/* Iterate over all reference chunks. */
		for( size_t uiChunkId = 0; uiChunkId < rxChunkedRef.uiNumberOfChunks; uiChunkId++ )
		{	/* Load the required reference segment into device */
			/* TO DO: We clear to many vectors over here. */
			this->clearDeviceVectors();

			/* Get the appropriate section of the reference in the device memory */
			metaMeasureAndLogDuration<false>
			(	"Time for loading reference",	// text message
				[&] () // lambda by reference
				{
					//// this->loadReference( *(rxChunkedRef.vChunks[uiChunkId]) );
					this->loadReference( *(rxChunkedRef.getChunk( uiChunkId ) ) );
				} // lambda
			); // function call

			/* Process the chunk via the kernel */
			metaMeasureAndLogDuration<false>
			(	"Time for kernel execution", // text message
				[&] () // lambda by reference
				{
					this->doQueryForChunk( xQueryProfile,
#if ( DO_TESTS == 1 )
												rvQuerySeq, // for test we require the original query sequence
#endif
												vHE_BackupVector );
				} // lambda
			); // function call

			/* Extract the maximum score and positions of the maximum score */
			this->xM_Vector.updateSharedBackup();
			
#if (USE_THREADPOOL == 1)
			if( this->xMaxExtractSuccess.valid() )
			{	/* The future is active and we have to block until 
				 * work of the previous maximum extraction is done 
				 */
				bool wait = this->xMaxExtractSuccess.get();
			} // if
			
			/* Set the future by queuing in the Threadpool */
			this->xMaxExtractSuccess
				= this->xThreadPool.enqueue( [&] ( size_t uiThreadId, size_t uiChunkIdLoc )
				{
					//// std::cout << "Thread " << uiThreadId << " starts max extraction" << std::endl;
					this->extractMaxima( rvMaxScorePositions,
										 iOverallMaxScore,
										 rxChunkedRef.numOfSymUsedInChunk( uiChunkIdLoc ),
										 uiChunkIdLoc * rxChunkedRef.uiChunkSize );
					//// std::cout << "Thread " << uiThreadId << " ends max extraction" << std::endl;
					return true;
				}, // lambda
				
				uiChunkId
			); // enqueue
#else			
			metaMeasureAndLogDuration<false>
			(	"Time for extracting maximum scores",	// text message
				[&] () // lambda by reference
				{
					this->extractMaxima( rvMaxScorePositions,
										 iOverallMaxScore,
										 rxChunkedRef.numOfSymUsedInChunk( uiChunkId ),
										 uiChunkId * rxChunkedRef.uiChunkSize );
				} // lambda
			); // function call
		
#endif
		} // for

#if (USE_THREADPOOL == 1)
		/* We have to wait until the final maximum extraction finished */
		if( this->xMaxExtractSuccess.valid() )
		{	/* The future is active and we have to block until work is done */
			//// std::cout << "WAIT FOR FINISHING MAX EXTRACTION" << std::endl;
			bool wait = this->xMaxExtractSuccess.get();
		} // if
		//// std::cout << "MAX EXTRACTION DONE" << std::endl;
#endif

		/* Sort the vector that comprises all maximum positions */
		std::sort( rvMaxScorePositions.begin(), rvMaxScorePositions.end() );
	} // method


	/* Collects the maxima contained in the maximum-vector.
	 * Optimization: Do the maximum collection of the previous kernel call during the next kernel call.
	 */
	void extractMaxima( std::vector<size_t> &rvMaxScorePositions, // vector keeping max positions
						SCORE_TP &iOverallMaxScore, // the maximum values
						const size_t uiNumOfSymUsedInChunk, // number of symbols actually used in chunk
						const size_t offset ) // reference offset for calculating the original position
	{
		assert( uiNumOfSymUsedInChunk > 0 && uiNumOfSymUsedInChunk <= this->uiRefCapicity );

		/* Fetch the ME-vector from the device */
		auto pME_VectorBackup = this->xM_Vector.getSharedBackup();
		auto &rvME_VectorBackup = *pME_VectorBackup;

		/* Find all maximum positions (done on CPU side).
		* (The maximum search could be done efficiently by a reduce kernel on GPU side)
		*/
		for( size_t uiItr = 0; uiItr < this->uiRefCapicity; uiItr++ )
		{
			if( rvME_VectorBackup[uiItr] >= iOverallMaxScore )
			{
				size_t uiSymPosInChunk = rowIdOptimizedToStandard( uiItr );
				if( uiSymPosInChunk >= uiNumOfSymUsedInChunk )
				{	/* Position is out of range */
					continue;
				} // if
				
				if( rvME_VectorBackup[uiItr] > iOverallMaxScore )
				{	// fresh overall maximum detected 
					rvMaxScorePositions.clear();
					iOverallMaxScore = rvME_VectorBackup[uiItr];
				} // if
#if	( OPTIMIZED_INDEXING == 0 )
				rvMaxScorePositions.push_back( this->rowIdOptimizedToStandard( uiRow ) );
#else
				//// rvMaxScorePositions.push_back( offset + uiItr );
				rvMaxScorePositions.push_back( offset + uiSymPosInChunk );
#endif // OPTIMIZED_INDEXING
			} // if
		} // for
	} // method

#if	( OPTIMIZED_INDEXING == 1 )
	/* Translates a optimized row coordinate to the real value */
	inline size_t rowIdOptimizedToStandard( size_t uiRowOptimized )
	{
		size_t uiSegmentPos = uiRowOptimized / this->uiNumberOfSegments;
		size_t uiSegmentId = uiRowOptimized % this->uiNumberOfSegments;
		return uiSegmentId * uiSegmentSize + uiSegmentPos;
	} // method

	/* Translates a real row coordinate to the optimized value */
	inline size_t rowIdStandardToOptimzied( size_t uiRowStandard )
	{
		size_t uiSegmentId = uiRowStandard % this->uiSegmentSize;
		size_t uiSegmentPos = uiRowStandard / this->uiSegmentSize;
		return uiSegmentId * uiNumberOfSegments + uiSegmentPos;
	} // method
#endif

#if ( DO_TESTS == 1 )
	/* Fetch a HF-vector from device */
	auto get_HF_Vector( bool bChooseVectorOne )
	{
		return bChooseVectorOne ? this->xHF_VectorOne.getCopyAsVector() : this->xHF_VectorTwo.getCopyAsVector();
	} // method

	void debugCheckHF_VectorTwo( std::shared_ptr<NaiveSW> pNaiveSW,
								 size_t uiColumn,
								 bool bChooseVectorOne )
	{
#if ( USE_PACKED == 1 )
		auto vDeviceVector = reinterpret_cast<std::vector<int2> &>(this->get_HF_Vector( bChooseVectorOne ));
		vDeviceVector = SequenceTransformer<int2>::inverseTransposedSeq( vDeviceVector, this->uiSegementSize );
		/* Doubles the size of the vector, so that it is equal to original reference size*/
		vDeviceVector = SequenceTransformer<int2>::unpackint2_2Vertically( vDeviceVector );
#else
		auto vDeviceVector = this->get_HF_Vector( bChooseVectorOne );
		assert( vDeviceVector.size() == CHUNK_SIZE );
		assert( this->uiRefCapicity == CHUNK_SIZE );
		vDeviceVector = SequenceTransformer<SCORE_TP2>::inverseTransposedSeq( vDeviceVector, this->uiSegementSize );
#endif

		auto vNaiveSW_HRow = pNaiveSW->getH_Row( uiColumn );
		for( size_t uiRow = 0; uiRow < this->uiRefCapicity; uiRow++ )
		{
#if	( OPTIMIZED_INDEXING == 0 )
			auto uiRowOnDeviceVector = this->rowIdStandardToOptimzied( uiRow );
#else
			auto uiRowOnDeviceVector = uiRow;
#endif // OPTIMIZED_INDEXING
			if( vDeviceVector[uiRowOnDeviceVector].x != vNaiveSW_HRow[uiRow] )
			{
				std::cout << "Diff at row " << uiRow << " column " << uiColumn << " value SW " 
					<< vNaiveSW_HRow[uiRow] << " value GPU " << static_cast<int>(vDeviceVector[uiRowOnDeviceVector].x)
					<< " relative_in_block " << (uiRow % this->uiSegementSize) << std::endl;
				exit( 0 );
			} // if
		} // for
	} // method
#endif // DO_TESTS

	/* Returns if the kernel in lazy mode could fix all inaccuracies.
	 * If we get true over here, the lazy kernel does not need to be called once more. 
	 */
	bool lazyFixed()
	{	/* Copy the lazy fixed-vector to host memory */
		this->pLazyFixedVector.copyToArray( this->pContinuationFlags.get() );
		auto pFlagsAsLong = this->pContinuationFlags.get();

		/* WARNING!: In order to get the below code working, the vector has to be defined of type long
		 * or you risk misalignments
		 */
		//// long* pFlagsAsLong = reinterpret_cast<long *>(this->pContinuationFlags.get());
		
		for( size_t uiCounter = 0; uiCounter < this->uiNumberOfSegments; ++uiCounter )
		{
			if( pFlagsAsLong[uiCounter] != 0 )
			{	/* Some segment indicates the need for recall of lazy-Checksum-loop */
				return false;
			} // if
		} // for
		return true;
	} // method	

	/* Destructor */
	~SW_GPU_Processor()
	{} // destructor
}; // class


/* Main function for ongoing CUDA work.
 * Returned value: Maximum score.
 * 32 bit scoring: <int4, int2, int, int>
 * 24 bit scoring: <float4, float2, float, float>
 */
template<typename SCORE_TP4, typename SCORE_TP2, typename SCORE_TP, typename CHECKSUM_TP>
std::vector<GPUReturn> cudaAlignTmpl
(   std::vector<char> &rvRefSeq, // reference sequence
	std::vector<std::vector<char>> &rvQuerySeqs, // vector of query sequences
    unsigned int uiDeviceId
) 
{	/* Do checks and reset device */
	assert( sizeof( SCORE_TP4 ) == 4 * sizeof( SCORE_TP ) && sizeof( SCORE_TP2 ) == 2 * sizeof( SCORE_TP ) );

    //// for(auto& rQ : rvQuerySeqs)
        for(char c :rvRefSeq)
            if(c >= 4)
                std::cout << "WARNING " << c << std::endl;
	
	hipSetDevice( uiDeviceId );
	/* Reset all GPU devices */
	hipDeviceReset();
	
	/* The raw size of the reference.*/
	size_t uiRefSize = rvRefSeq.size();

	size_t uiRefChunkCapicity = CHUNK_SIZE; // depends on the memory available
	SW_GPU_Processor<SCORE_TP4, SCORE_TP2, SCORE_TP, CHECKSUM_TP, TILE_WIDTH> xSW_GPU_Processor
	(	uiDeviceId, // device id 
		uiRefChunkCapicity ); // capacity for single reference chunk
	
	/* Set the segment size according to the freshly loaded reference.
	 * (At this point we know that the reference-size is a multiple of the number of segments.)
	 * TO DO: Set this as part of the GPU initialization.
	 */
	xSW_GPU_Processor.uiSegmentSize = CHUNK_SIZE / xSW_GPU_Processor.uiNumberOfSegments; 

	/* Create the segmented reference.
	 * For queries against the same reference this has to be done only once!
	 */
	ChunkedTransposedReference xChunkedReference( rvRefSeq, // reference sequence
												  uiRefChunkCapicity, // primary segment size (should be equal to strip capacity)
												  xSW_GPU_Processor.uiNumberOfSegments ); // secondary segment size
	
    std::vector<GPUReturn> vRet; 
    //std::vector<std::pair<SCORE_TP, std::vector<size_t>>> vRet;

	for( auto &rvQuerySeq : rvQuerySeqs )
    {
        SCORE_TP iOverallMaxScore = 0;
        std::vector<size_t> vMaxScorePositions;

	    /* Do the core alignment */
	    metaMeasureAndLogDuration<false>
	    (	"GPU time",	// logging text
	    	[&] () // lambda by reference
	    	{
	    		xSW_GPU_Processor.doQueryForChunkedReference
	    		( rvQuerySeq, // query
	    		  xChunkedReference, // reference in segmented form
	    		  vMaxScorePositions, // vector that receives maximum positions
	    		  iOverallMaxScore ); // overall maximum score
	    	} // lambda
	    ); // function call

        //// std::cout << "iOverallMaxScore: " << iOverallMaxScore << std::endl;
        vRet.emplace_back( iOverallMaxScore, vMaxScorePositions );
    } // for (iterating queries)

	return vRet;
} // function

size_t getNumberOfCards()
{
	int iDeviceCount; 

	hipGetDeviceCount( &iDeviceCount );
	CUERR

	return iDeviceCount;
} // function


std::vector<GPUReturn> cudaAlign
(
    std::vector<char> &rvRefSeq, // reference sequence
	std::vector<std::vector<char>> &rvQuerySeqs, // vector of query sequences
    unsigned int uiDeviceId
)
{
    return cudaAlignTmpl<int4, int2, int, int>(rvRefSeq, rvQuerySeqs, uiDeviceId);
};

#if 0
std::vector<GPUReturn> cudaAlignPool
(
	std::vector<char> &rvRefSeq, // reference sequence
	std::vector<std::vector<char>> &rvQuerySeqs // vector of query sequences
)
{
	auto uiNumberOfCards = getNumberOfCards();
	std::cout << "Number of cards: " << uiNumberOfCards << std::endl;

	ThreadPool xThreadPool(uiNumberOfCards);

	std::vector< std::future<GPUReturn> > results;

	for ( size_t i = 0; i < rvQuerySeqs.size(); i++ )
	{	/* i must be passed by value into the lambda */
		results.push_back( xThreadPool.enqueue(
			[i] ( size_t uiThreadId )
			{
				cudaAlign()
			//// std::cout << "Thread " << uiThreadId << " starts max extraction" << std::endl;
				this->extractMaxima( rvMaxScorePositions,
									 iOverallMaxScore,
									 rxChunkedRef.numOfSymUsedInChunk( uiChunkIdLoc ),
									 uiChunkIdLoc * rxChunkedRef.uiChunkSize );
				//// std::cout << "Thread " << uiThreadId << " ends max extraction" << std::endl;
				return true;
			} // lambda
		)); 
		
			for ( size_t i = 0; i < results.size(); ++i )
				BOOST_LOG_TRIVIAL( trace ) << "*** See result for " << i << " as " << results[i].get();
		// std::cout << results[i].get() << ' ';



		xThreadPool.enqueue( [i] ( size_t id, int j ) { itemWorker( i, id, j ); }, 6 );
	} // for

	this->xMaxExtractSuccess
		

									 uiChunkId
		); // enqueue

	std::future<GPUReturn> xMaxExtractSuccess;


	
	// return cudaAlignTmpl<int4, int2, int, int>(rvRefSeq, rvQuerySeqs);
	return std::vector<GPUReturn>();
};
#endif
